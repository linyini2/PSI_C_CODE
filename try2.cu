#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <string>
#include <random>
#include <ctime>
#include <chrono>
#include <typeinfo>
#include "error.cuh"
#include "hipfft/hipfft.h"
#include <hipfft/hipfftXt.h>
using namespace std;

static const int64_t _two31 = INT64_C(1) << 31; // 2^31
static const int64_t _two32 = INT64_C(1) << 32; // 2^32
typedef uint32_t Torus32;

const int BLOCK_SIZE = 1;
const int GRID_SIZE = 1;

class Params128
{
public:
    int N;
    int n;
    int bk_l;
    int bk_Bgbit;
    float ks_stdev;
    float bk_stdev;
    int Bg;
    int Msize;
    double *H;
    uint32_t inter;

public:
    Params128(int N, int n, int bk_l, int bk_Bgbit, float ks_stdev, float bk_stdev, int Msize) : N(N),
                                                                                                 n(n),
                                                                                                 bk_l(bk_l),
                                                                                                 bk_Bgbit(bk_Bgbit),
                                                                                                 ks_stdev(ks_stdev),
                                                                                                 bk_stdev(bk_stdev),
                                                                                                 Bg(pow(2, bk_Bgbit)),
                                                                                                 Msize(Msize),
                                                                                                 inter(_two31 / Msize * 2)
    {
        this->H = (double *)malloc(sizeof(double) * this->bk_l);
        for (int i = 0; i < this->bk_l; i++)
        {
            this->H[i] = pow(this->Bg, (-(i + 1)));
            // cout << this->H[i] << endl;
        }
    }
    ~Params128() {
        free(H);
    }
};

// Params128& getp128()
// {
//     // Params128 *p128 = new Params128(4, 630, 2, 10, pow(2.0, -15.4), pow(2.0, -28), 2);
//     return Params128(4, 630, 2, 10, pow(2.0, -15.4), pow(2.0, -28), 2);
// }

int sign(float d)
{
    if (d > 0)
    {
        return 1;
    }
    else if (d < 0)
    {
        return -1;
    }
    else
    {
        return 0;
    }
}

uint32_t mutoT(int mu, int Msize)
{
    return uint32_t(_two31 / Msize * 2 * mu);
}

Torus32 dtot32(float d)
{
    int dsign = sign(d);
    return Torus32(round(fmod(d * dsign, 1) * _two32) * dsign);
}

void gaussian32(uint32_t *vecmu, float alpha, uint32_t *h_ga, int size = 1)
{
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine gen(seed);
    std::normal_distribution<double> dis(0, alpha);

    for (int i = 0; i < size; i++)
    {
        h_ga[i] = dtot32(dis(gen)) + vecmu[i];
    }
}

uint32_t Ttomu(uint32_t phase, uint32_t inter)
{
    uint32_t half = uint32_t(inter / 2);
    return uint32_t(uint32_t(phase + half) / inter);
}

int32_t* trlweKeyGen(Params128 p128)
{
    int32_t *trlwekey = (int32_t *)malloc(p128.N * sizeof(int32_t));
    srand((int)time(NULL));
    for (int i = 0; i < p128.N; i++)
    {
        trlwekey[i] = rand() % 2;
    }
    return trlwekey;
}

__global__ void product(hipDoubleComplex *d_Comp_a, hipDoubleComplex *d_Comp_trlwekey, hipDoubleComplex *d_Comp_product, Params128 p128)
{
    int M = p128.N / 2;
    for (int i = 0; i < M; i++)
    {
        d_Comp_product[i] = hipCmul(d_Comp_a[i], d_Comp_trlwekey[i]);
    }
}

uint32_t *PolyMul(uint32_t *h_a, int32_t *h_trlwekey, Params128 p128)
{
    int M = p128.N / 2;
    hipfftHandle plan;
    hipfftPlan1d(&plan, M, HIPFFT_Z2Z, 1);

    // process h_a
    hipDoubleComplex *h_Comp_a = (hipDoubleComplex *)malloc(M * sizeof(hipDoubleComplex));
    hipDoubleComplex *d_Comp_a;
    hipMalloc((void **)&d_Comp_a, M * sizeof(hipDoubleComplex));

    for (int i = 0; i < M; i++)
    {
        h_Comp_a[i].x = h_a[i];
        h_Comp_a[i].y = h_a[i + M];
    }
    hipMemcpy(d_Comp_a, h_Comp_a, M * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipfftExecZ2Z(plan, (hipDoubleComplex *)d_Comp_a, (hipDoubleComplex *)d_Comp_a, HIPFFT_FORWARD);
    hipDeviceSynchronize();

    // process h_trlwekey
    hipDoubleComplex *h_Comp_trlwekey = (hipDoubleComplex *)malloc(M * sizeof(hipDoubleComplex));
    hipDoubleComplex *d_Comp_trlwekey;
    hipMalloc((void **)&d_Comp_trlwekey, M * sizeof(hipDoubleComplex));

    for (int i = 0; i < M; i++)
    {
        h_Comp_trlwekey[i].x = h_trlwekey[i];
        h_Comp_trlwekey[i].y = h_trlwekey[i + M];
    }
    hipMemcpy(d_Comp_trlwekey, h_Comp_trlwekey, M * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipfftExecZ2Z(plan, (hipDoubleComplex *)d_Comp_trlwekey, (hipDoubleComplex *)d_Comp_trlwekey, HIPFFT_FORWARD);
    hipDeviceSynchronize();

    // process mul
    hipDoubleComplex *h_Comp_product = (hipDoubleComplex *)malloc(p128.N * sizeof(hipDoubleComplex));
    hipDoubleComplex *d_Comp_product;
    hipMalloc((void **)&d_Comp_product, M * sizeof(hipDoubleComplex));

    product<<<GRID_SIZE, BLOCK_SIZE>>>(d_Comp_a, d_Comp_trlwekey, d_Comp_product, p128);

    hipfftExecZ2Z(plan, (hipDoubleComplex *)d_Comp_product, (hipDoubleComplex *)d_Comp_product, HIPFFT_BACKWARD);
    hipDeviceSynchronize();
    hipMemcpy(h_Comp_product, d_Comp_product, M * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    uint32_t *h_product = (uint32_t *)malloc(p128.N * sizeof(uint32_t));
    for (int i = 0; i < M; i++)
    {
        h_product[i] = h_Comp_product[i].x;
        h_product[i + M] = h_Comp_product[i].y;
    }

    hipfftDestroy(plan);
    free(h_Comp_a);
    free(h_Comp_trlwekey);
    free(h_Comp_product);
    hipFree(d_Comp_a);
    hipFree(d_Comp_trlwekey);
    hipFree(d_Comp_product);

    return h_product;
}

uint32_t **trlweSymEnc(uint32_t *vecmu, int32_t *trlwekey, Params128 p128)
{
    uint32_t *b = (uint32_t *)malloc(p128.N * sizeof(uint32_t));
    uint32_t *a = (uint32_t *)malloc(p128.N * sizeof(uint32_t));

    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::mt19937 g(seed);
    for (int i = 0; i < p128.N; i++)
    {
        a[i] = g();
    }

    uint32_t *product = (uint32_t *)malloc(p128.N * sizeof(uint32_t));
    product = PolyMul(a, trlwekey, p128);

    uint32_t *ga = (uint32_t *)malloc(p128.N * sizeof(uint32_t));
    gaussian32(vecmu, p128.ks_stdev, ga, p128.N);

    for (int i = 0; i < p128.N; i++)
    {
        b[i] = (ga[i] - product[i]) % (_two32);
    }

    uint32_t **c = (uint32_t **)malloc(2 * sizeof(uint32_t *));
    for (int i = 0; i < 2; i++)
    {
        c[i] = (uint32_t *)malloc(p128.N * sizeof(uint32_t));
    }
    for (int i = 0; i < p128.N; i++)
    {
        c[0][i] = b[i];
    }
    for (int i = 0; i < p128.N; i++)
    {
        c[1][i] = a[i];
    }

    // for (int i = 0; i < 2; i++)
    // {
    //     for (int j = 0; j < p128.N; j++)
    //     {
    //         cout << c[i][j] << " ";
    //     }
    //     cout <<  endl;
    // }

    free(b);
    free(a);
    free(product);
    free(ga);

    return c;
}

uint32_t *trlweSymDec(uint32_t **c, int32_t *trlwekey, Params128 p128)
{
    uint32_t *product = (uint32_t *)malloc(p128.N * sizeof(uint32_t));
    product = PolyMul(c[1], trlwekey, p128);

    uint32_t *phase = (uint32_t *)malloc(p128.N * sizeof(uint32_t));
    for (int i = 0; i < p128.N; i++)
    {
        phase[i] = c[0][i] + product[i];
    }

    uint32_t *mu = (uint32_t *)malloc(p128.N * sizeof(uint32_t));
    for (int i = 0; i < p128.N; i++)
    {
        mu[i] = Ttomu(phase[i], p128.inter);
    }

    free(product);
    free(phase);

    return mu;
}

uint32_t*** trgswSymEnc(uint32_t *vecmu, int32_t *trlwekey, Params128 p128)
{
    // uint32_t **muh = (uint32_t **)malloc(p128.bk_l * sizeof(uint32_t *));
    // for (int i = 0; i < p128.bk_l; i++)
    // {
    //     muh[i] = (uint32_t *)malloc(p128.N * sizeof(uint32_t));
    // }
    // cout << p128.Bg << endl;
    // cout << p128.bk_Bgbit << endl;
    // for (int j = 0; j < p128.bk_l; j++)
    // {
    //     cout << p128.H[j] << "  ";
    // }
    // external product : H * plain
    // for (int i = 0; i < p128.bk_l; i++)
    // {
    //     cout << p128.H[i] << "----" << endl;
        // for (int j = 0; j < p128.N; j++)
        // {
        //     cout << vecmu[j] << "----";
        //     muh[i][j] = p128.H[i] * vecmu[j];
        //     cout << muh[i][j] << "~~~~";
            
        // }
        // cout << endl;
    // }

    int lines = 2 * p128.bk_l;
    uint32_t ***c;
    c = (uint32_t ***)malloc(lines * sizeof(uint32_t **));

    for (int i = 0; i < lines; i++)
    {
        c[i] = (uint32_t **)malloc(2 * sizeof(uint32_t *));
    }
    for (int i = 0; i < lines; i++)
    {
        for (int j = 0; j < 2; j++)
        {
            c[i][j] = (uint32_t *)malloc(p128.N * sizeof(uint32_t));
        }
    }

    // uint32_t *vec_zero = (uint32_t *)malloc(sizeof(uint32_t) * p128.N);
    // for (int i = 0; i < p128.N; i++)
    // {
    //     vec_zero[i] = 0;
    // }

    // for (int i = 0; i < lines; i++)
    // {
    //     c[i] = trlweSymEnc(vec_zero, trlwekey, p128);
    // }
    // for (int i = 0; i < p128.bk_l; i++)
    // {
    //     for (int j = 0; j < p128.N; j++)
    //     {
    //         c[i][0][j] += muh[i][j];
    //     }
    // }
    // for (int i = p128.bk_l; i < lines; i++)
    // {
    //     for (int j = 0; j < p128.N; j++)
    //     {
    //         c[i][1][j] += muh[i][j];
    //     }
    // }

    // free operations
    // for (int i = 0; i < p128.bk_l; i++)
    // {
    //     free(muh[i]);
    // }
    // free(muh);

    return c;
}

uint32_t* trgswSymDec(uint32_t ***c, int32_t *trlwekey, Params128 p128)
{
    uint32_t *phase = (uint32_t *)malloc(p128.N * sizeof(uint32_t));
    uint32_t *product = (uint32_t *)malloc(p128.N * sizeof(uint32_t));
    uint32_t *vecmu = (uint32_t *)malloc(p128.N * sizeof(uint32_t));
    product = PolyMul(c[0][1], trlwekey, p128);
    for (int i = 0; i < p128.N; i++)
    {
        phase[i] = (c[0][0][i] + product[i]) * p128.Bg;
        vecmu[i] = Ttomu(phase[i], p128.inter);
        cout << vecmu[i] << endl;
    }
    free(phase);
    free(product);
    return vecmu;
}

uint32_t*** fun(uint32_t *vecmu, int32_t *trlwekey, Params128 p128)
{
    uint32_t **muh = (uint32_t **)malloc(p128.bk_l * sizeof(uint32_t *));
    for (int i = 0; i < p128.bk_l; i++)
    {
        muh[i] = (uint32_t *)malloc(p128.N * sizeof(uint32_t));
    }

    for (int i = 0; i < p128.bk_l; i++)
    {
        for (int j = 0; j < p128.N; j++)
        {
            muh[i][j] = p128.H[i] * vecmu[j];
            cout << muh[i][j] << "~~~~";
        }
        cout << endl;
    }

    int lines = 2 * p128.bk_l;
    uint32_t ***c;
    c = (uint32_t ***)malloc(lines * sizeof(uint32_t **));

    for (int i = 0; i < lines; i++)
    {
        c[i] = (uint32_t **)malloc(2 * sizeof(uint32_t *));
    }
    for (int i = 0; i < lines; i++)
    {
        for (int j = 0; j < 2; j++)
        {
            c[i][j] = (uint32_t *)malloc(p128.N * sizeof(uint32_t));
        }
    }

    uint32_t *vec_zero = (uint32_t *)malloc(sizeof(uint32_t) * p128.N);
    for (int i = 0; i < p128.N; i++)
    {
        vec_zero[i] = mutoT(1, p128.Msize);
        cout << vec_zero[i] << endl;
    }
    uint32_t **cipher = (uint32_t **)malloc(2 * sizeof(uint32_t *));
    for (int i = 0; i < 2; i++)
    {
        cipher[i] = (uint32_t *)malloc(p128.N * sizeof(uint32_t));
    }
    cipher = trlweSymEnc(vec_zero, trlwekey, p128);
    uint32_t *mu = (uint32_t *)malloc(sizeof(uint32_t) * p128.N);
    trlweSymDec(cipher, trlwekey, p128);
    for (int i = 0; i < p128.N; i++)
    {
        cout << mu[i] << endl;
    }
    
    // for (int i = 0; i < 2; i++)
    // {
    //     for (int j = 0; j < p128.N; j++)
    //     {
    //         cout << c[i][j] << " ";
            
    //     }
    //     cout << "------------------" << endl;
    // }

    // for (int i = 0; i < lines; i++)
    // {
    //     c[i] = trlweSymEnc(vec_zero, trlwekey, p128);
    // }
    // for (int i = 0; i < lines; i++)
    // {
    //     for (int j = 0; j < 2; j++)
    //     {
    //         for (int k = 0; k < p128.N; k++)
    //         {
    //             cout << c[i][j][k] << " ";
    //         }   
    //         cout << endl;
    //     }
    //     cout << "------------------" << endl;
    // }


    for (int i = 0; i < p128.bk_l; i++)
    {
        free(muh[i]);
    }
    free(muh);
    free(mu);

    return c;
}

void Test()
{
    Params128 p128 = Params128(4, 630, 2, 10, pow(2.0, -15.4), pow(2.0, -28), 2);

    uint32_t *vecmu = (uint32_t *)malloc(p128.N * sizeof(uint32_t));
    for (int i = 0; i < p128.N / 2; i++)
    {
        if (i % 2 == 0)
        {
            vecmu[i] = mutoT(1, p128.Msize);
        }
        else
        {
            vecmu[i] = mutoT(1, p128.Msize);
        }
    }
    for (int i = p128.N / 2; i < p128.N; i++)
    {
        vecmu[i] = mutoT(0, p128.Msize);
    }

    // trlwekey generation
    int32_t *trlwekey = (int32_t *)malloc(p128.N * sizeof(int32_t));
    trlwekey = trlweKeyGen(p128);

    int lines = 2 * p128.bk_l;
    uint32_t ***c;
    c = (uint32_t ***)malloc(lines * sizeof(uint32_t **));

    for (int i = 0; i < lines; i++)
    {
        c[i] = (uint32_t **)malloc(2 * sizeof(uint32_t *));
    }
    for (int i = 0; i < lines; i++)
    {
        for (int j = 0; j < 2; j++)
        {
            c[i][j] = (uint32_t *)malloc(p128.N * sizeof(uint32_t));
        }
    }
    c = fun(vecmu, trlwekey, p128);

    free(vecmu);
    free(trlwekey);
    for (int i = 0; i < lines; i++)
    {
        for (int j = 0; j < 2; j++)
        {
            free((c[i][j]));
        }
    }
    for (int i = 0; i < p128.N; i++)
    {
        free(c[i]);
    }
}

void Test_TRLWE()
{
    Params128 p128 = Params128(4, 630, 2, 10, pow(2.0, -15.4), pow(2.0, -28), 2);

    uint32_t *vecmu = (uint32_t *)malloc(p128.N * sizeof(uint32_t));
    for (int i = 0; i < p128.N; i++)
    {
        if (i % 2 == 0)
        {
            vecmu[i] = mutoT(1, p128.Msize);
        }
        else
        {
            vecmu[i] = mutoT(0, p128.Msize);
        }
    }
    for (int i = 0; i < p128.N; i++)
    {
        cout << vecmu[i] << endl;
    }
    cout << "-----------------" << endl;
    // trlwekey generation
    int32_t *trlwekey = (int32_t *)malloc(p128.N * sizeof(int32_t));
    trlwekey = trlweKeyGen(p128);

    uint32_t **cipher = (uint32_t **)malloc(2 * sizeof(uint32_t *));
    for (int i = 0; i < 2; i++)
    {
        cipher[i] = (uint32_t *)malloc(p128.N * sizeof(uint32_t));
    }
    cipher = trlweSymEnc(vecmu, trlwekey, p128);
    uint32_t *mu = (uint32_t *)malloc(sizeof(uint32_t) * p128.N);
    mu = trlweSymDec(cipher, trlwekey, p128);
    for (int i = 0; i < p128.N; i++)
    {
        cout << mu[i] << endl;
    }

    free(vecmu);
    free(trlwekey);
    for (int i = 0; i < p128.N; i++)
    {
        free(cipher[i]);
    }
    free(cipher);
    free(mu);
}

int main()
{
    Test_TRLWE();
    return 0;
}