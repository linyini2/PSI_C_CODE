#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <string>
#include <random>
#include <ctime>
#include <chrono>
#include<typeinfo>
#include "error.cuh"
#include "hipfft/hipfft.h"
#include <hipfft/hipfftXt.h>
using namespace std;

static const int64_t _two31 = INT64_C(1) << 31; // 2^31
static const int64_t _two32 = INT64_C(1) << 32; // 2^32
typedef uint32_t Torus32;
const int N = 4;
const int M = N / 2;

const int Msize = 2;
const int BLOCK_SIZE = 1;
const int GRID_SIZE = 1;
const uint32_t inter = uint32_t(_two31 / Msize * 2);
float alpha = pow(2.0, -15.4);


int sign(float d)
{
    if (d > 0)
    {
        return 1;
    }
    else if (d < 0)
    {
        return -1;
    }
    else
    {
        return 0;
    }
}

uint32_t mutoT(int mu, int Msize)
{
    return uint32_t(_two31 / Msize * 2 * mu);
}

Torus32 dtot32(float d)
{
    int dsign = sign(d);
    return Torus32(round(fmod(d * dsign, 1) * _two32) * dsign);
}

void gaussian32(uint32_t *vecmu, float alpha, uint32_t *h_ga, int size = 1)
{
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine gen(seed);
    std::normal_distribution<double> dis(0, alpha);

    for (size_t i = 0; i < size; i++)
    {
        h_ga[i] = dtot32(dis(gen)) + vecmu[i];
        // h_ga[i].x = dtot32(dis(gen)) + vecmu[i];
        // h_ga[i].y = 0;
    }
}

uint32_t Ttomu(uint32_t phase, uint32_t inter)
{
    uint32_t half = uint32_t(inter / 2);
    return uint32_t(uint32_t(phase + half) / inter);
}

void trlweKeyGen(int32_t *h_trlwekey)
{
    cout << "htrlwekey: " << endl;
    srand((int)time(NULL));
    for (int i = 0; i < N; i++)
    {
        h_trlwekey[i] = rand() % 2;
        cout << h_trlwekey[i] << endl;
    }
}

__global__ void product(hipDoubleComplex* d_Comp_a, hipDoubleComplex* d_Comp_trlwekey, hipDoubleComplex* d_Comp_product)
{
	for (size_t i = 0; i < M; i++)
    {
        /* code */
        d_Comp_product[i] = hipCmul(d_Comp_a[i], d_Comp_trlwekey[i]);
    }
}

void PolyMul(uint32_t *h_a, int32_t *h_trlwekey, uint32_t *h_product)
{
    hipfftHandle plan;
    hipfftPlan1d(&plan, M, HIPFFT_Z2Z, 1);
    
    // process h_a
    hipDoubleComplex *h_Comp_a = (hipDoubleComplex *)malloc(M * sizeof(hipDoubleComplex));
    hipDoubleComplex *d_Comp_a;
    hipMalloc((void **)&d_Comp_a, M * sizeof(hipDoubleComplex));

    for (int i = 0; i < M; i++)
    {
        h_Comp_a[i].x = h_a[i];
        h_Comp_a[i].y = h_a[i + M];
    }
    hipMemcpy(d_Comp_a, h_Comp_a, M * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipfftExecZ2Z(plan, (hipDoubleComplex *)d_Comp_a, (hipDoubleComplex *)d_Comp_a, HIPFFT_FORWARD);
    hipDeviceSynchronize();
    hipMemcpy(h_Comp_a, d_Comp_a, M * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    cout  << "h_Comp_a: polynomial Point value representation" << endl;
    for (int i = 0; i < M; i++)
    {
        cout << i << ": " << h_Comp_a[i].x << ", " << h_Comp_a[i].y << endl;
    }
    cout << "------------------" << endl;
    
    // process h_trlwekey
    hipDoubleComplex *h_Comp_trlwekey = (hipDoubleComplex *)malloc(M * sizeof(hipDoubleComplex));
    hipDoubleComplex *d_Comp_trlwekey;
    hipMalloc((void **)&d_Comp_trlwekey, M * sizeof(hipDoubleComplex));

    for (int i = 0; i < M; i++)
    {
        h_Comp_trlwekey[i].x = h_trlwekey[i];
        h_Comp_trlwekey[i].y = h_trlwekey[i + M];
    }
    hipMemcpy(d_Comp_trlwekey, h_Comp_trlwekey, M * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipfftExecZ2Z(plan, (hipDoubleComplex *)d_Comp_trlwekey, (hipDoubleComplex *)d_Comp_trlwekey, HIPFFT_FORWARD);
    hipDeviceSynchronize();
    hipMemcpy(h_Comp_trlwekey, d_Comp_trlwekey, M * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    cout  << "h_Comp_trlwekey: polynomial Point value representation" << endl;
    for (int i = 0; i < M; i++)
    {
        cout << i << ": " << h_Comp_trlwekey[i].x << ", " << h_Comp_trlwekey[i].y << endl;
    }
    cout << "------------------" << endl;

    // process mul
    hipDoubleComplex *h_Comp_product = (hipDoubleComplex *)malloc(N * sizeof(hipDoubleComplex));
    hipDoubleComplex *d_Comp_product;
    hipMalloc((void **)&d_Comp_product, M * sizeof(hipDoubleComplex));

    product<<<GRID_SIZE,BLOCK_SIZE>>>(d_Comp_a, d_Comp_trlwekey, d_Comp_product);
    hipMemcpy(h_Comp_product, d_Comp_product, M * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    cout  << "polynomial mul :Point value representation" << endl;
    for (int i = 0; i < M; i++)
    {
        cout << i << ": " << h_Comp_product[i].x << ", " << h_Comp_product[i].y << endl;
    }
    cout << "------------------" << endl;

    hipfftExecZ2Z(plan, (hipDoubleComplex *)d_Comp_product, (hipDoubleComplex *)d_Comp_product, HIPFFT_BACKWARD);
    hipDeviceSynchronize();
    hipMemcpy(h_Comp_product, d_Comp_product, M * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    cout  << "h_product:" << endl;
    for (int i = 0; i < M; i++)
    {
        h_product[i] = h_Comp_product[i].x;
        h_product[i + M] = h_Comp_product[i].y;
    }
    
    for (int i = 0; i < M; i++)
    {
        cout << i << ": " << h_product[i] << endl;
    }

    hipfftDestroy(plan);
    free(h_Comp_a);
    free(h_Comp_trlwekey);
    free(h_Comp_product);
    hipFree(d_Comp_a);
    hipFree(d_Comp_trlwekey);
    hipFree(d_Comp_product);
}

void TwistFFT(uint32_t *a, hipDoubleComplex *CompData)
{
    for (int i = 0; i < N; i++)
    {
        CompData[i].x = a[i];
        CompData[i].y = 0;
    }
    
    hipDoubleComplex *d_CompData;
    hipMalloc((void **)&d_CompData, N * sizeof(hipDoubleComplex));
    hipMemcpy(d_CompData, CompData, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    hipfftHandle plan;
    hipfftPlan1d(&plan, N, HIPFFT_Z2Z, 1);
    hipfftExecZ2Z(plan, (hipDoubleComplex *)d_CompData, (hipDoubleComplex *)d_CompData, HIPFFT_FORWARD);
    hipDeviceSynchronize();
    hipMemcpy(CompData, d_CompData, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(d_CompData);
}

void TwistIFFT(hipDoubleComplex *CompData)
{
    hipDoubleComplex *d_CompData;
    hipMalloc((void **)&d_CompData, N * sizeof(hipDoubleComplex));
    hipMemcpy(d_CompData, CompData, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    hipfftHandle plan;
    hipfftPlan1d(&plan, N, HIPFFT_Z2Z, 1);
    hipfftExecZ2Z(plan, (hipDoubleComplex *)d_CompData, (hipDoubleComplex *)d_CompData, HIPFFT_BACKWARD);
    hipDeviceSynchronize();
    hipMemcpy(CompData, d_CompData, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(d_CompData);
}


void trlweSymEnc(uint32_t *vecmu, float alpha, int32_t *h_trlwekey, uint32_t *cipher, uint32_t *h_a)
{
    // uint32_t *h_a = (uint32_t *)malloc(N * sizeof(uint32_t));

    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::mt19937 g(seed);
    cout << "h_a:" << endl;
    for (int i = 0; i < N; i++)
    {
        h_a[i] = g();
        cout << h_a[i] << endl;
    }

    uint32_t *h_product = (uint32_t *)malloc(N * sizeof(uint32_t));
    PolyMul(h_a, h_trlwekey, h_product);
    
    uint32_t *h_ga = (uint32_t *)malloc(N * sizeof(uint32_t));
    gaussian32(vecmu, alpha, h_ga, N);
    cout << "h_ga:" << endl;
    for (int i = 0; i < N; i++)
    {
        cout << h_ga[i] << endl;
    }

    cout << "cipher:" << endl;
    for (int i = 0; i < N; i++)
    {
        // cipher[i] = round(h_ga[i] - h_product[i]) % (_two32);
        cipher[i] = (h_ga[i] - h_product[i]) % (_two32);
        cout << cipher[i] << endl;
    }

    free(h_product);
    free(h_ga);
}

void trlweSymDec(uint32_t *cipher, uint32_t *h_a, int32_t *h_trlwekey)
{
    uint32_t *h_product = (uint32_t *)malloc(N * sizeof(uint32_t));
    PolyMul(h_a, h_trlwekey, h_product);

    cout << "dec phase = mu + e" << endl;
    uint32_t *h_phase = (uint32_t *)malloc(N * sizeof(uint32_t));
    for (int i = 0; i < N; i++)
    {
        h_phase[i] = cipher[i] + h_product[i];
    }
    cout << "decryption result: " << endl;
    for (int i = 0; i < N; i++)
    {
        cout << i << ": " << Ttomu(h_phase[i], inter) << endl;
    }

}

void Test()
{
    // generate message
    cout << "message to Torus: " << endl;
    uint32_t *vecmu = (uint32_t *)malloc(N * sizeof(uint32_t));
    for (int i = 0; i < N; i++)
    {
        if(i % 2 == 0)
        {
            vecmu[i] = mutoT(0, Msize);
        }
        else
        {
            vecmu[i] = mutoT(1, Msize);
        }
        cout << vecmu[i] << endl;
    }

    cout << "---------------------------------" << endl;
    // trlwekey generation
    int32_t *h_trlwekey = (int32_t *)malloc(N * sizeof(int32_t));
    trlweKeyGen(h_trlwekey);

    // encryption
    uint32_t *cipher = (uint32_t *)malloc(N * sizeof(uint32_t));
    uint32_t *a = (uint32_t *)malloc(N * sizeof(uint32_t));
    trlweSymEnc(vecmu, alpha, h_trlwekey, cipher, a);

    // decryption
    trlweSymDec(cipher, a, h_trlwekey);

    free(vecmu);
    free(h_trlwekey);
    free(cipher);
    free(a);
}


int main()
{
    Test();
    return 0;
}